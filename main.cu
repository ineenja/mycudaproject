#include <vector>
#include <iostream>

//#include "modulation/modulation.cuh"
//#include "demodulation/demodulation.cuh"
//#include "myparser/myparser.cuh"
#include "filtering/filtering.cuh"

int main() {

    int m = 5, n = 5;
    std::vector<float> inputSignal = {1, 0, 0 ,0 ,0};
    std::vector<float> outputSignal(inputSignal.size());
    std::vector<float> numerator = {2, 2, 3 ,1 ,2};
    std::vector<float> denumerator = {1, -1, 0.1, 2, 0.1};
    std::vector<float> memory(inputSignal.size(), 0.0);
    int order = 5;

    float *inputSignalPtr, *outputSignalPtr;
    float *numeratorPtr, *denumeratorPtr;
    float *memoryPtr;

    hipMalloc(&inputSignalPtr, order * sizeof(float ));
    hipMalloc(&outputSignalPtr, order * sizeof(float ));
    hipMalloc(&numeratorPtr, n * sizeof(float ));
    hipMalloc(&denumeratorPtr, m * sizeof(float ));
    hipMalloc(&memoryPtr, order * sizeof(float ));

    hipMemcpy(inputSignalPtr, inputSignal.data(), order * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(outputSignalPtr, outputSignal.data(), order * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(numeratorPtr, numerator.data(), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(denumeratorPtr, denumerator.data(), m * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(memoryPtr, memory.data(), order * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (order + blockSize - 1) / blockSize;
    filterKernel<<<gridSize, blockSize>>>(inputSignalPtr, outputSignalPtr, order, numeratorPtr, denumeratorPtr, memoryPtr, order);

    hipMemcpy(outputSignal.data(), outputSignalPtr, order * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(inputSignalPtr);
    hipFree(outputSignalPtr);
    hipFree(numeratorPtr);
    hipFree(denumeratorPtr);
    hipFree(memoryPtr);

    for (float i : outputSignal){
        std::cout << i << " ";
    }

    return 0;
}